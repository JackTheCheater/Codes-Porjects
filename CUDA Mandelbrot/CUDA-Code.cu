
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#define RESOLUTION 1000
#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)
#define STEP ((double) RATIO_X / WIDTH)

// Maximum number of iterations
#define ITERATIONS 100

using namespace std;

__global__ void mandelbrot_kernel(int *image)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < HEIGHT && col < WIDTH)
	{
        double x = col * STEP + MIN_X;
        double y = row * STEP + MIN_Y;

        double real = 0.0;
        double imag = 0.0;

		int count = 0;

		for (int i = 0; i < ITERATIONS; i++)
		{
			double new_real = real * real - imag * imag + x;
            double new_imag = 2.0 * real * imag + y;
            real = new_real;
            imag = new_imag;

			//If it is convergent
			if (real * real + imag * imag >= 4.0)
			{
				break;
			}

			count++;
		}

		image[row * WIDTH + col] = count;
	}
}

int main(int argc, char **argv)
{
	int threads = atoi(argv[1]);

	const auto start = chrono::steady_clock::now();

	int *cuda_image;
	hipMallocManaged(&cuda_image, WIDTH * HEIGHT * sizeof(int*));

	dim3 block_size(threads);
	dim3 image_size((int)(HEIGHT * WIDTH + block_size.x) / block_size.x);

	mandelbrot_kernel <<<image_size, block_size>>> (cuda_image);

	hipDeviceSynchronize();

	const auto end = chrono::steady_clock::now();
	
	cout << "Time elapsed: " << chrono::duration_cast<chrono::milliseconds> (end - start).count() << " milliseconds." << endl;

	// Write the result to a file
	ofstream matrix_out;

	matrix_out.open("Matrix-file-cuda.txt", ios::trunc);
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -2;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << cuda_image[row * WIDTH + col];

			if (col < WIDTH - 1)
				matrix_out << ',';
		}

		if (row < HEIGHT - 1)
			matrix_out << endl;
	}

	matrix_out.close();

	hipFree(cuda_image);
	return 0;
}
